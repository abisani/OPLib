
#include <hip/hip_runtime.h>
// sgemv4.cu --- Part of the project OPLib 1.0, a high performance pricing library
// based on operator methods, higher level BLAS and multicore architectures 

// Author:     2009 Claudio Albanese
// Maintainer: Claudio Albanese <claudio@albanese.co.uk>
// Created:    April-July 2009
// Version:    1.0.0
// Credits:    The CUDA code for SGEMM4, SGEMV4 and SSQMM were inspired by 
//             Vasily Volkov's implementation of SGEMM
//			   We use several variations of the multi-threaded Mersenne Twister algorithm of 
//			   period 2203 due to Makoto Matsumoto.
//             The Monte Carlo routine in SMC includes code by Victor Podlozhnyuk 
//             included in the CUDA SDK.
//             CPU-side BLAS and random number generators link to primitives in the
//			   Intel Math Kernel Libraries. 

// This program is free software; you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation; either version 2 of the License, or
// (at your option) any later version.
// 
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with this program; see the file COPYING.  If not, write to
// the Free Software Foundation, Inc., 59 Temple Place - Suite 330,
// Boston, MA 02111-1307, USA.



#define NCOLS 4


#ifdef LINUX
#define __declspec(x)
#define __stdcall
#endif


__device__ void rank1_update( const float a, const float *b, float *c )
{
	c[0] += a*b[0];
	c[1] += a*b[1];
	c[2] += a*b[2];
	c[3] += a*b[3];
	c[4] += a*b[4];
	c[5] += a*b[5];
	c[6] += a*b[6];
	c[7] += a*b[7];
	c[8] += a*b[8];
	c[9] += a*b[9];
	c[10] += a*b[10];
	c[11] += a*b[11];
	c[12] += a*b[12];
	c[13] += a*b[13];
	c[14] += a*b[14];
	c[15] += a*b[15];
}

__device__ void rankk_update( int k, const float *A0, int lda, const float *b, int ldb, float *c )
{
    if( k <= 0 ) return;
    const float *A = A0;  

    int i = 0;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c ); if( ++i >= k ) return; A += lda;
    rank1_update( A[0], &b[i*ldb], c );
}


static __global__ void global_sgemv4(unsigned int * argptr_bid)
{

__shared__ unsigned int arg[10];

	const unsigned int * argptr = (unsigned int *) argptr_bid[blockIdx.x];	
	
	if(threadIdx.x<=9) arg[threadIdx.x] = argptr[threadIdx.x];
    __syncthreads();
	
	const int blockIdx_x = arg[1];
	const int blockIdx_y = arg[2];
	const int d = arg[3];
	const int nz = arg[4];
	float * A = (float *)(arg[6]);
	float * B = (float *)(arg[7]);
	const int * col0_v = (int *)(arg[8]);
	const int * col1_v = (int *)(arg[9]);
	

	const int ibx = blockIdx_x * 64;
	const int iby = blockIdx_y * 16;
	const int row = ibx + threadIdx.x + threadIdx.y * 16;
	const int twonzldc = 2 * nz * d;
	int k = d;

	A += row;
	float *C = B + row; 
	B += threadIdx.x; 
  
	float c[16] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};

	__shared__ int scol0_v[16];		
	__shared__ int scol1_v[16];		

	if(threadIdx.y == 0) scol0_v[threadIdx.x] = col0_v[iby + threadIdx.x];
	if(threadIdx.y == 1) scol1_v[threadIdx.x] = col1_v[iby + threadIdx.x];
	__syncthreads();

		    
	__shared__ float b[16][17];
	
	for( ; k > 0; k -= 16 )
	{
		b[threadIdx.x][threadIdx.y]  = B[scol0_v[threadIdx.y]];
		b[threadIdx.x][threadIdx.y + 4]  = B[scol0_v[threadIdx.y + 4]];
		b[threadIdx.x][threadIdx.y + 8]  = B[scol0_v[threadIdx.y + 8]];
		b[threadIdx.x][threadIdx.y + 12] = B[scol0_v[threadIdx.y + 12]];
	__syncthreads();
		
		if( k < 16 )  break;

		#pragma unroll
	    for( int i = 0; i < 16; i++, A += d )  rank1_update( A[0], &b[i][0], c ); 
	    __syncthreads();
		B += 16;
	};

    rankk_update( k, A, d, &b[0][0], 17, c );

    if( row >= d ) return;

	int col1;
	col1 = scol1_v[0]; if(col1 >= twonzldc )  return; C[col1] = c[0]; 
	col1 = scol1_v[1]; if(col1 >= twonzldc )  return; C[col1] = c[1]; 
    col1 = scol1_v[2]; if(col1 >= twonzldc )  return; C[col1] = c[2]; 
    col1 = scol1_v[3]; if(col1 >= twonzldc )  return; C[col1] = c[3]; 
    col1 = scol1_v[4]; if(col1 >= twonzldc )  return; C[col1] = c[4]; 
    col1 = scol1_v[5]; if(col1 >= twonzldc )  return; C[col1] = c[5]; 
    col1 = scol1_v[6]; if(col1 >= twonzldc )  return; C[col1] = c[6]; 
    col1 = scol1_v[7]; if(col1 >= twonzldc )  return; C[col1] = c[7]; 
    col1 = scol1_v[8]; if(col1 >= twonzldc )  return; C[col1] = c[8]; 
    col1 = scol1_v[9]; if(col1 >= twonzldc )  return; C[col1] = c[9]; 
    col1 = scol1_v[10]; if(col1 >= twonzldc ) return; C[col1] = c[10]; 
    col1 = scol1_v[11]; if(col1 >= twonzldc ) return; C[col1] = c[11]; 
    col1 = scol1_v[12]; if(col1 >= twonzldc ) return; C[col1] = c[12]; 
    col1 = scol1_v[13]; if(col1 >= twonzldc ) return; C[col1] = c[13]; 
    col1 = scol1_v[14]; if(col1 >= twonzldc ) return; C[col1] = c[14]; 
    col1 = scol1_v[15]; if(col1 >= twonzldc ) return; C[col1] = c[15]; 
    
	};



extern "C" void __declspec( dllexport ) opcuda_sgemv4(int nblocks, unsigned argptr_bid)
{	

  dim3 grid(nblocks, 1), threads( 16, 4 );
  global_sgemv4<<<grid, threads>>>((unsigned int *) argptr_bid);   

}	



